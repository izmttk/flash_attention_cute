#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cute/tensor.hpp>
#include <iostream>
#include <chrono>
#include <random>

template <class Layout>
__host__ __device__ void my_print_layout(Layout const& layout)  // (m,n) -> idx
{
  using namespace cute;
//   CUTE_STATIC_ASSERT_V(rank(layout) == Int<2>{});

  int idx_width = num_digits(cosize(layout)) + 2;

  print(layout); print("\n");

  // Column indices
  print("    ");
  for (int n = 0; n < size<1>(layout); ++n) { printf("  %*d ", idx_width-2, n); }
  printf("\n");

  // Print out A m-by-n
  for (int m = 0; m < size<0>(layout); ++m) {
    // Header
    print("    ");
    for (int n = 0; n < size<1>(layout); ++n) {
      printf("+");
      for (int i = 0; i < idx_width; ++i) { printf("-"); }
    }
    printf("+\n");
    // Values
    printf("%2d  ", m);  // Row indices
    for (int n = 0; n < size<1>(layout); ++n) { printf("| %*d ", idx_width-2, int(layout(m,n))); }
    printf("|\n");
  }
  // Footer
  print("    ");
  for (int n = 0; n < size<1>(layout); ++n) {
    printf("+");
    for (int i = 0; i < idx_width; ++i) { printf("-"); }
  }
  printf("+\n");
}

__global__ void gemm_device(
    float *Aptr,
    float *Bptr,
    float *Cptr,
    int M,
    int N,
    int K
) {
    constexpr int kTileM = 128;
    constexpr int kTileN = 128;
    constexpr int kTileK = 8;
    using namespace cute;

    // mA: M x K col-major
    // mB: N x K col-major
    // mC: M x N col-major
    Tensor mA = make_tensor(make_gmem_ptr(Aptr), make_shape(M, K), make_stride(Int<1>{}, M));
    Tensor mB = make_tensor(make_gmem_ptr(Bptr), make_shape(N, K), make_stride(Int<1>{}, N));
    Tensor mC = make_tensor(make_gmem_ptr(Cptr), make_shape(M, N), make_stride(Int<1>{}, M));


    // gA: (kTileM, kTileK, k) col-major
    // gB: (kTileN, kTileK, k) col-major
    // gC: (kTileM, kTileN) col-major
    Tensor gA = local_tile(mA, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(blockIdx.y, _));
    Tensor gB = local_tile(mB, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(blockIdx.x, _));
    Tensor gC = local_tile(mC, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(blockIdx.y, blockIdx.x));

    // predicate tensor, 元素值是坐标元组，用来检查 gmem 坐标是否越界
    Tensor idA = make_identity_tensor(shape(mA));
    Tensor idB = make_identity_tensor(shape(mB));
    Tensor idC = make_identity_tensor(shape(mC));
    // 对 idA idB 采取相同的切分方式
    Tensor cA = local_tile(idA, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(blockIdx.y, _));
    Tensor cB = local_tile(idB, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(blockIdx.x, _));
    Tensor cC = local_tile(idC, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(blockIdx.y, blockIdx.x));


    auto sA_layout = make_layout(make_shape(Int<kTileM>{}, Int<kTileK>{})); // m-major, 即 stride m = 1, 即 col-major
    auto sB_layout = make_layout(make_shape(Int<kTileN>{}, Int<kTileK>{})); // n-major, 即 stride m = 1, 即 col-major
    
    __shared__ float smemA[cosize(sA_layout)];
    __shared__ float smemB[cosize(sB_layout)];
    
    Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout);
    Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout);

    // 这意味着 (32, 8) 个线程，每个线程 copy (4, 1) 个元素，并且最小操作单位是 uint128_t，即 4 个 float
    // TiledCopy 规定了执行一次 TiledCopy 操作处理的数据布局/规模，包括每个线程需要负责哪些数据
    TiledCopy copy_A = make_tiled_copy(
        Copy_Atom<UniversalCopy<uint128_t>, float>{}, // copy_atom(copy_op, copy_type)
        make_layout(make_shape(Int<32>{}, Int<8>{})), // Thread Layout (32, 8)
        make_layout(make_shape(Int<4>{}, Int<1>{}))   // Value Layout (4, 1)
    );
    // 根据 Thread Layout 通过 threadIdx.x 索引到线程负责的数据块
    // ThrCopy 规定了当前线程需要负责的数据
    ThrCopy thr_copy_A = copy_A.get_slice(threadIdx.x);

    // kThrM = 4, kThrN = 4, kThrK = 1
    // partition_S|D 的结果是 ((CPY...), 按 TiledCopy 将 gA|B 分块的坐标...)
    // CPY: (CopyAtomValLayout, ValLayout / CopyAtomValLayout)，即一个 ValLayout 的大小
    // TiledCopy 的分块坐标可能少于 gA|B 的维度，只分块前几个维度，后面维度尺寸会保持不变
    Tensor tAgA = thr_copy_A.partition_S(gA); // (CPY, 1, 1, k) = ((4, 1), 1, 1, k)
    Tensor tAsA = thr_copy_A.partition_D(sA); // (CPY, 1, 1)
    Tensor tAcA = thr_copy_A.partition_S(cA); // (CPY, 1, 1, k)
    Tensor tApA = make_tensor_like<bool>(tAsA(0, _, _));


    TiledCopy copy_B = make_tiled_copy(
        Copy_Atom<UniversalCopy<uint128_t>, float>{}, // copy_atom(copy_op, copy_type)
        make_layout(make_shape(Int<32>{}, Int<8>{})), // Thread Layout (32, 8)
        make_layout(make_shape(Int<4>{}, Int<1>{}))   // Value Layout (4, 1)
    );
    ThrCopy thr_copy_B = copy_B.get_slice(threadIdx.x);
    Tensor tBgB = thr_copy_B.partition_S(gB); // (CPY, 1, 1, k)
    Tensor tBsB = thr_copy_B.partition_D(sB); // (CPY, 1, 1)
    Tensor tBcB = thr_copy_B.partition_S(cB); // (CPY, 1, 1, k)
    Tensor tBpB = make_tensor_like<bool>(tBsB(0, _, _));

    // 1x1x1 FMA atom, repeat 16x16x1
    // TiledMMA 规定了执行一次 TiledMMA 操作处理的数据布局/规模，包括每个线程需要负责哪些数据
    TiledMMA mma = make_tiled_mma(
        MMA_Atom<UniversalFMA<float>>{},  // mma_atom(mma_op) 1x1x1
        make_layout(make_shape(Int<16>{}, Int<16>{}, Int<1>{})) // Thread Layout (16, 16, 1)
    );
    // 这意味着我们一个 block 中所有线程一次要处理的问题规模为 MxNxK = 16x16x1
    // 子问题 C = A * B, C(16, 16), A(16, 1), B(1, 16)
    // make_tiled_mma 还可以有第三个参数 PermutationMNK, 用于表示对 M N K 维度的置换
    // 用这个参数可以实现一个线程多次计算 MMA_Atom，如何做？？？

    // 别的教程可能这里会使用 get_thread_slice, 其实是一个函数
    ThrMMA thr_mma = mma.get_slice(threadIdx.x);
    // MMA 表示 一次 MMA_Atom 消耗的数据量/布局，MMA_M/N 就是被分解的子问题布局了
    // TiledMMA 定义的子问题规模为 MNK = 16x16x1，那么 sA 被 16x1 分块，sB 被 1x16 分块，gC 被 16x16 分块
    // 每个线程需要在这些分块上重复计算，具体来说 sA|B(128, 8) 被分为 (8, 8) 块，gC(128, 128) 被分为 (8, 8) 块
    // 那么每个线程需要串行循环计算 8x8x8 个 MMA_Atom
    // partition_A|B|C 的结果是 ((单个MMA_Op消耗的数据布局...), 按 TiledMMA 将 sA|sB|gC 分块的坐标...)
    Tensor tCsA = thr_mma.partition_A(sA); // (MMA, MMA_M, MMA_K)
    Tensor tCsB = thr_mma.partition_B(sB); // (MMA, MMA_N, MMA_K)
    Tensor tCgC = thr_mma.partition_C(gC); // (MMA, MMA_M, MMA_N)
    Tensor tCcC = thr_mma.partition_C(cC);

    Tensor tArA = make_fragment_like(tAsA);
    Tensor tBrB = make_fragment_like(tBsB);

    Tensor tCrA = thr_mma.make_fragment_A(tCsA); // (MMA, MMA_M, MMA_K)
    Tensor tCrB = thr_mma.make_fragment_B(tCsB); // (MMA, MMA_N, MMA_K)
    // 这里是按照 tCgC 的布局来创建一个 Tensor
    Tensor tCrC = thr_mma.make_fragment_C(tCgC); // (MMA ,MMA_M, MMA_N)

    // SM80 以下的 GPU，从 gmem 复制到 smem 实际上是 LDG + STS，需要使用中间寄存器

    // 对 K 维度的分块计算进行 pipeline 优化
    // 未使用 pipeline 的指令流
    //                                  v sync                                      v sync     v sync
    // +---------------------+----------+----------+----------+---------------------+----------+----------+----------+
    // | LDG T1              | STS T1   | LDS T1   | MMA T1   | LDG T2              | STS T2   | LDS T2   | MMA T2   | ...
    // +---------------------+----------+----------+----------+---------------------+----------+----------+----------+
    //
    // 使用 pipeline(two-stage, double-buffer) 的指令流
    //                                  v sync                v sync     v sync                v sync     v sync
    // +---------------------+----------+---------------------+----------+---------------------+----------+
    // | LDG T1              | STS T1   | LDG T2              | STS T2   | LDG T3              | STS T3   | ...
    // +---------------------+----------+----------+----------+----------+----------+----------+----------+
    //                                  | LDS T1   | MMA T1   |          | LDS T2   | MMA T2   |
    //                                  +----------+----------+          +----------+----------+
    //                       ^~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~^ 循环节
    //                                  ^~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~^ 或者这种循环节也可以，但是不高效
    // 注意指令发射的顺序，MMA T1 在访存指令之后，才能保证 T2 访存和 T1 计算并行化
    // 这里由于直接使用 tAsA 和 tBsB 作为 gemm 参数，LDS 和 MMA 我们不能分离，因此 LDS 和 MMA 放在 LDG 之后
    // 即 gemm 要在调用读取全局内存之后再调用，还要确保当前阶段的所有 LDS 在下一阶段的 STS 之前调用，因为每个阶段复用同一区域 smem
    // 
    // 现在更进一步，每一个分块都包含了多个 MMA，我们再按照 K 维度分块这些 MMA，并行 LDS 和 MMA 执行
    //                                  v sync                                      v sync     v sync
    // +---------------------+----------+          +---------------------+          +----------+          +---------------------+
    // | LDG T1              | STS T1   |          | LDG T2              |          | STS T2   |          | LDG T3              | ...
    // +---------------------+----------+----------+----------+----------+----------+----------+----------+----------+----------+
    //                                  | LDS T1.1 | LDS T1.2 | LDS T1.3 | LDS T1.4 |          | LDS T2.1 | LDS T2.2 | LDS T2.3 |
    //                                  +----------+----------+----------+----------+          +----------+----------+----------+
    //                                             | MMA T1.1 | MMA T1.2 | MMA T1.3 |          | MMA T1.4 | MMA T2.1 | MMA T2.2 |
    //                                             +----------+----------+----------+          +----------+----------+----------+
    //                                             ^~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~^ 循环节


    // 先将 k_tile = 0 的块的寄存器、共享内存完成读取
    copy(copy_A, tAgA(_, _, _, 0), tArA);
    copy(copy_B, tBgB(_, _, _, 0), tBrB);
    
    clear(tCrC); // 放在这里是为了可以与上面的 copy 重叠

    copy(tArA, tAsA);
    copy(tBrB, tBsB);
    __syncthreads();

    // 加载第一个 tile 第一块的寄存器数据
    copy(tCsA(_, _, 0), tCrA(_, _, 0));
    copy(tCsB(_, _, 0), tCrB(_, _, 0));

    auto K_TILE_MAX = size<3>(tAgA);
    auto K_BLOCK_MAX = size<2>(tCrC);
    for (int k_tile = 0; k_tile < K_TILE_MAX; k_tile++) {
        CUTE_UNROLL
        for (int k_block = 0; k_block < K_BLOCK_MAX; k_block++) {
            // 如果是当前 tile 的最后一块，将保存全局内存数据的寄存器写回 smem
            if (k_block + 1 == K_BLOCK_MAX) {
                if (k_tile  < K_TILE_MAX - 1) {
                    __syncthreads();
                    copy(tArA, tAsA);
                    copy(tBrB, tBsB);
                    __syncthreads();
                }
            }
            // 读取下一个块的数据到寄存器
            int k_block_next = (k_block + 1) % K_BLOCK_MAX;
            copy(tCsA(_, _, k_block_next), tCrA(_, _, k_block_next));
            copy(tCsB(_, _, k_block_next), tCrB(_, _, k_block_next));
            // 如果是当前 tile 的第一块，开始下一个 tile 全局内存读取到寄存器
            if (k_block == 0) {
                if (k_tile  < K_TILE_MAX - 1) {
                    copy(copy_A, tAgA(_, _, _, k_tile + 1), tArA);
                    copy(copy_B, tBgB(_, _, _, k_tile + 1), tBrB);
                }
            }
            // gemm
            gemm(mma, tCrA(_, _, k_block), tCrB(_, _, k_block), tCrC);
        }
    }

    CUTE_UNROLL
    for (int i = 0; i < size(tCrC); i++) {
        if (elem_less(tCcC(i), make_coord(M, N))) {
            tCgC(i) = tCrC(i); // 没用 axpby, 因为直接计算的 C = A * B
        }
    }
}

void launch_gemm(
    float *Aptr,
    float *Bptr,
    float *Cptr,
    int M,
    int N,
    int K
) {
    auto cdiv = [](int a, int b) { return (a + b - 1) / b; };
    dim3 grid(cdiv(N, 128), cdiv(M, 128));
    dim3 block(256);
    gemm_device<<<grid, block>>>(Aptr, Bptr, Cptr, M, N, K);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
}

void gemm_host(
    float *Aptr,
    float *Bptr,
    float *Cptr,
    int M,
    int N,
    int K
) {
    // gemm nt, A(M, K) col-major, B(N, K) col-major, C(M, N) col-major
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float sum = 0;
            for (int k = 0; k < K; k++) {
                sum += Aptr[k * M + m] * Bptr[k * N + n];
            }
            Cptr[n * M + m] = sum;
        }
    }
}


float *generate_data(int n) {
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    auto distribution = std::uniform_real_distribution<float>(1.0, 10.0);
    float *data = new float[n];
    for (int i = 0; i < n; i++) {
        data[i] = distribution(generator);
    }
    return data;
}

int main() {
    // 注意由于使用了向量化访存，数据应当对齐 16 字节，即 M 是 4 的倍数
    const int M = 4, N = 4, K = 17;
    float *a_h = generate_data(M * K);
    float *b_h = generate_data(K * N);

    float *out_d, *a_d, *b_d;
    hipMalloc(&out_d, sizeof(float) * M * N);
    hipMalloc(&a_d, sizeof(float) * M * K);
    hipMalloc(&b_d, sizeof(float) * K * N);
    hipMemcpy(a_d, a_h, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(float) * K * N, hipMemcpyHostToDevice);

    namespace chrono = std::chrono;
    chrono::time_point<chrono::high_resolution_clock> start, end;
    chrono::duration<double, std::milli> elapsed;

    start = chrono::high_resolution_clock::now();
    launch_gemm(a_d, b_d, out_d, M, N, K);
    hipDeviceSynchronize();
    end = chrono::high_resolution_clock::now();
    elapsed = end - start;
    std::cout << "Calculation time: " << elapsed.count() << " ms" << std::endl;

    float* out_h = (float *)malloc(sizeof(float) * M * N);
    hipMemcpy(out_h, out_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    float* out_hh = (float *)malloc(sizeof(float) * M * N);
    gemm_host(a_h, b_h, out_hh, M, N, K);

    printf("out_h: \n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", out_h[j * M + i]);
        }
        printf("\n");
    }

    printf("out_hh: \n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", out_hh[j * M + i]);
        }
        printf("\n");
    }
    return 0;
}
